#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__constant__ BYTE gpu_difficulty_5_zeros[SHA256_HASH_SIZE] = "0000099999999999999999999999999999999999999999999999999999999999";


__global__ void findNonce(BYTE *block_content, int current_length, BYTE *block_hash, bool *found, uint64_t *nonce) {
    int local_nonce = blockIdx.x * blockDim.x + threadIdx.x;
    if (local_nonce > MAX_NONCE) {
        return;
    }
    if (*found) {
        return;
    }

    char local_nonce_string[NONCE_SIZE];
    BYTE local_block_hash[SHA256_HASH_SIZE];

    intToString(local_nonce, local_nonce_string);
    d_strcpy((char *)block_content + current_length, local_nonce_string);
    apply_sha256(block_content, d_strlen((const char *)block_content), local_block_hash, 1);

    if (*found == false && compare_hashes(local_block_hash, gpu_difficulty_5_zeros) <= 0) {
        d_strcpy((char *)block_hash, (const char *)local_block_hash);
        *found = true;
        *nonce = local_nonce;
    }
}

int main(int argc, char **argv) {
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
        tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
        tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
    BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
    uint64_t nonce = 0;
    size_t current_length;

    // Top hash
    apply_sha256(tx1, strlen((const char *)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char *)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char *)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char *)tx4), hashed_tx4, 1);
    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char *)tx12), hashed_tx12, 1);
    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char *)tx34), hashed_tx34, 1);
    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char *)tx34), top_hash, 1);

    // prev_block_hash + top_hash
    strcpy((char *)block_content, (const char *)prev_block_hash);
    strcat((char *)block_content, (const char *)top_hash);
    current_length = strlen((char *)block_content);

    hipEvent_t start, stop;
    startTiming(&start, &stop);

    int block_size = 128;
    int nr_blocks = MAX_NONCE / block_size;

    BYTE *d_block_content, *d_block_hash;
    bool *d_found;
    uint64_t *d_nonce;

    hipMalloc((void **)&d_block_content, BLOCK_SIZE);
    hipMalloc((void **)&d_block_hash, SHA256_HASH_SIZE);
    hipMalloc((void **)&d_found, sizeof(bool));
    hipMalloc((void **)&d_nonce, sizeof(uint64_t));

    hipMemcpy(d_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
    {
        bool found = false;
        hipMemcpy(d_found, &found, sizeof(bool), hipMemcpyHostToDevice);
    }

    findNonce << <nr_blocks, block_size >> > (d_block_content, current_length, d_block_hash, d_found, d_nonce);

    float seconds = stopTiming(&start, &stop);

    hipMemcpy(block_hash, d_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(&nonce, d_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);

    printf("Time: %f\n", seconds);

    printResult(block_hash, nonce, seconds);

    hipFree(d_block_content);
    hipFree(d_block_hash);
    hipFree(d_found);
    hipFree(d_nonce);

    return 0;
}
