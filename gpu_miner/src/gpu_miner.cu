#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__constant__ BYTE gpu_difficulty_5_zeros[SHA256_HASH_SIZE] = "0000099999999999999999999999999999999999999999999999999999999999";


__global__ void findNonce(BYTE *block_content, int current_length, BYTE *block_hash, bool *found, uint64_t *nonce) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    uint64_t l_start = thread_id * MAX_NONCE / total_threads;
    uint64_t l_end = (thread_id + 1) * MAX_NONCE / total_threads;

    char local_nonce_string[NONCE_SIZE];
    BYTE local_block_hash[SHA256_HASH_SIZE];
    BYTE local_block_content[BLOCK_SIZE];
    d_strcpy((char *)local_block_content, (const char *)block_content);

    for (uint64_t local_nonce = l_start; *found != true && local_nonce < l_end; local_nonce++) {
        intToString(local_nonce, local_nonce_string);
        d_strcpy((char *)local_block_content + current_length, local_nonce_string);
        apply_sha256(local_block_content, d_strlen((const char *)local_block_content), local_block_hash, 1);

        if (*found == false && compare_hashes(local_block_hash, gpu_difficulty_5_zeros) <= 0) {
            d_strcpy((char *)block_hash, (const char *)local_block_hash);
            *found = true;
            *nonce = local_nonce;
        }
    }
}

int cuda_get_sm_count() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return prop.multiProcessorCount;
}

int main(int argc, char **argv) {
    BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
        tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
        tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
    BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
    uint64_t nonce = 0;
    size_t current_length;

    // Top hash
    apply_sha256(tx1, strlen((const char *)tx1), hashed_tx1, 1);
    apply_sha256(tx2, strlen((const char *)tx2), hashed_tx2, 1);
    apply_sha256(tx3, strlen((const char *)tx3), hashed_tx3, 1);
    apply_sha256(tx4, strlen((const char *)tx4), hashed_tx4, 1);
    strcpy((char *)tx12, (const char *)hashed_tx1);
    strcat((char *)tx12, (const char *)hashed_tx2);
    apply_sha256(tx12, strlen((const char *)tx12), hashed_tx12, 1);
    strcpy((char *)tx34, (const char *)hashed_tx3);
    strcat((char *)tx34, (const char *)hashed_tx4);
    apply_sha256(tx34, strlen((const char *)tx34), hashed_tx34, 1);
    strcpy((char *)tx1234, (const char *)hashed_tx12);
    strcat((char *)tx1234, (const char *)hashed_tx34);
    apply_sha256(tx1234, strlen((const char *)tx34), top_hash, 1);

    // prev_block_hash + top_hash
    strcpy((char *)block_content, (const char *)prev_block_hash);
    strcat((char *)block_content, (const char *)top_hash);
    current_length = strlen((char *)block_content);

    hipEvent_t start, stop;
    startTiming(&start, &stop);

    int block_size = 256;
    int nr_blocks = cuda_get_sm_count();

    BYTE *d_block_content, *d_block_hash;
    bool *d_found;
    uint64_t *d_nonce;

    hipMalloc((void **)&d_block_content, BLOCK_SIZE);
    hipMalloc((void **)&d_block_hash, SHA256_HASH_SIZE);
    hipMalloc((void **)&d_found, sizeof(bool));
    hipMalloc((void **)&d_nonce, sizeof(uint64_t));

    hipMemcpy(d_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
    {
        bool found = false;
        hipMemcpy(d_found, &found, sizeof(bool), hipMemcpyHostToDevice);
    }

    findNonce << <nr_blocks, block_size >> > (d_block_content, current_length, d_block_hash, d_found, d_nonce);

    float seconds = stopTiming(&start, &stop);

    hipMemcpy(block_hash, d_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(&nonce, d_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);

    printResult(block_hash, nonce, seconds);

    hipFree(d_block_content);
    hipFree(d_block_hash);
    hipFree(d_found);
    hipFree(d_nonce);

    return 0;
}
